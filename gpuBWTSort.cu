#include "hip/hip_runtime.h"

#define BZ_GPU
#include "bzlib_private.h"
#include <stdio.h>
#include <algorithm>
#include <fstream>
#include <iterator>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/scatter.h>
#include <thrust/iterator/zip_iterator.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define MAX_THREADS_PER_BLOCK 1024

//#define __DEBUG__

__device__ unsigned int *global_int_original_string = NULL; 
__device__ unsigned int *global_first_sort_rank = NULL;

class Bar
{
	unsigned int *functor_string;
	unsigned int *functor_first_sort_rank;
	const int currentOffset;
	const int currentLength;  
	const int totalLength; 

	public:
		__host__
		Bar(int _currentOffset, int _currentLength, int _totalLength):functor_string(global_int_original_string), functor_first_sort_rank(global_first_sort_rank), currentOffset(_currentOffset), currentLength(_currentLength), totalLength(_totalLength) { }
	
		inline __device__
		bool operator() (thrust::tuple< unsigned int, unsigned int > t1, thrust::tuple< unsigned int, unsigned int > t2) { 
			int seg1 = thrust::get<0>(t1); 
			int seg2 = thrust::get<0>(t2);
			
			if(seg1 > seg2) return false;
			if(seg1 < seg2) return true;
			
			int ind1 = thrust::get<1>(t1);
			int ind2 = thrust::get<1>(t2);
			int count = 0;
			
			while( count < currentLength) {
				int newInd1 = ( ind1 + currentOffset + count ) %totalLength;
				int newInd2 = ( ind2 + currentOffset + count ) %totalLength;
				unsigned int a1 = (functor_string[newInd1]); 
				unsigned int a2 = (functor_string[newInd2]);
				int a = a1 - a2; 
				count+=4;
				if( a == 0 ) { continue; }
				else {
					if((a1>>24) > (a2>>24)) { return false;}
					else if((a1>>24) < (a2>>24)) {return true;} 
					else if((a1>>16) > (a2>>16)) { return false;}
					else if((a1>>16) < (a2>>16)) {return true;} 
					else if((a1>>8) > (a2>>8)) { return false;}
					else if((a1>>8) < (a2>>8)) {return true;} 
					else if((a1) > (a2)) { return false;}
					else if((a1) < (a2)) {return true;} 
				}
			}
			return false;
		}
};

__global__ void pack4CharsToInt(unsigned char *input_string, unsigned int *static_input_string, unsigned int *output_string, unsigned int *output_index, unsigned int *d_array_first_sort_rank, int length) { 
	
	int threadID = (blockIdx.x * blockDim.x) +  threadIdx.x;

	if(threadID > length) return;

	d_array_first_sort_rank[threadID] = 0;
	if(threadID % 3 != 0) { 
		int mult3 =(int) ((1.0*threadID)/3);
	
		int newIndex = mult3*2 + ((threadID % 3) - 1);

		output_index[newIndex] = threadID;
	
		output_string[newIndex] = (((unsigned int)input_string[threadID]) << 24) + 
					  (((unsigned int)input_string[(threadID+1) % length]) << 16) +  
	                                  (((unsigned int)input_string[(threadID+2) % length]) << 8) + 
					  (((unsigned int)input_string[(threadID+3) % length]));
	}

	static_input_string[threadID] =  (((unsigned int)input_string[threadID]) << 24) + 
					 (((unsigned int)input_string[(threadID+1) % length]) << 16) +  
	                                 (((unsigned int)input_string[(threadID+2) % length]) << 8) + 
					 (((unsigned int)input_string[(threadID+3) % length]));

}

__global__ void findSuccessor( unsigned int *d_array_original_string, unsigned int *d_array_string, unsigned int *d_array_index, unsigned int *d_array_segment, unsigned int *d_array_string_out, unsigned int *d_array_segment_out, int length, int originalLength, int sequenceCount) {

	int threadID = (blockIdx.x * blockDim.x) +  threadIdx.x;
	if(threadID > length) return;
	d_array_segment_out[threadID] = 0; 
	if(threadID > 0) { 
		if(((d_array_string[threadID]!=d_array_string[threadID-1]) || (d_array_segment[threadID]!=d_array_segment[threadID-1])) ) { 
			d_array_segment_out[threadID] = 1; 
		}
	}
	int successorIndex = (d_array_index[threadID] + sequenceCount + 4)%originalLength; 
	d_array_string_out[threadID] = d_array_original_string[successorIndex];
}


__global__ void  eliminateSizeOneKernel1(unsigned int *d_array_original_string, unsigned int *d_array_final_index, unsigned int *d_array_index, unsigned int *d_array_static_index, unsigned int *d_array_map, unsigned int *d_array_stencil, unsigned int *d_array_first_sort_rank, int sequenceCount, int length, int originalLength) {

	int threadID = (blockIdx.x * blockDim.x) +  threadIdx.x;

	if(threadID >= length) return;

	d_array_stencil[threadID] = 1;

        if(threadID == 0 && (d_array_map[threadID + 1] == 1)) { 
		d_array_stencil[threadID] = 0; 
	} else if( (threadID == (length-1)) && (d_array_map[threadID] == 1) ) {
		d_array_stencil[threadID] = 0;  
	} else if( (d_array_map[threadID] == 1) && (d_array_map[threadID + 1] == 1)) { 
		d_array_stencil[threadID] = 0; 
	}

	if(d_array_stencil[threadID] == 0) {
		int finalIndex = d_array_index[threadID];
		d_array_final_index[ d_array_static_index[threadID] ] = finalIndex;
		d_array_first_sort_rank[finalIndex] = d_array_static_index[threadID];
	}
}

__global__ void updateSegments( unsigned int *d_int_array_string, unsigned int *d_array_index, unsigned int *d_array_segment, unsigned int *d_array_segment_out, int size, int offset, int length, int originalLength) {

	int threadID = (blockIdx.x * blockDim.x) +  threadIdx.x;
	if(threadID > size) return;
	d_array_segment_out[threadID] = 0; 
	if(threadID > 0) { 
		if((d_array_segment[threadID - 1] != d_array_segment[threadID])) { 
			d_array_segment_out[threadID] = 1;
			return; 
		}
		int count = 0;
		unsigned int ind1 = d_array_index[threadID - 1]; 
		unsigned int ind2 = d_array_index[threadID]; 
		while(count < length) {
			if( d_int_array_string[(ind1 + offset + count) % originalLength] != d_int_array_string[(ind2 + offset + count) % originalLength]) { 
				d_array_segment_out[threadID] = 1; 
				break;
			}
			count+=4; 
		} 
	}
	return;
}


__global__ void  eliminateSizeOne(unsigned int *d_array_final_index, unsigned int *d_array_index, unsigned int *d_array_static_index, unsigned int *d_array_map, unsigned int *d_array_stencil, unsigned int *d_array_first_sort_rank, int size, int originalLength) {

	int threadID = (blockIdx.x * blockDim.x) +  threadIdx.x;

	if(threadID >= size) return;

	d_array_stencil[threadID] = 1;

	if(threadID == 0 && (d_array_map[threadID + 1] == 1)) { 
		d_array_stencil[threadID] = 0; 
	} else if( (threadID == (size-1)) && (d_array_map[threadID] == 1) ) {
		d_array_stencil[threadID] = 0;  
	} else if( (d_array_map[threadID] == 1) && (d_array_map[threadID + 1] == 1)) { 
		d_array_stencil[threadID] = 0; 
	}

	if(d_array_stencil[threadID] == 0) {
		int finalIndex = d_array_index[threadID];
		d_array_final_index[ d_array_static_index[threadID] ] = finalIndex;
		d_array_first_sort_rank[finalIndex] = d_array_static_index[threadID]; 
	}
}

__global__ void createSecondSort(unsigned char *d_original_string_in, unsigned int *d_array_first_sort_rank, unsigned char *d_array_second_sort, unsigned int *d_array_second_sort_rank, unsigned int *d_array_second_sort_index, int secondSortLength) { 

	int threadID = (blockIdx.x * blockDim.x) +  threadIdx.x;

	if(threadID > secondSortLength) return;

	int mult3 = threadID*3;

	d_array_second_sort[threadID] = d_original_string_in[mult3];
	d_array_second_sort_rank[threadID] = d_array_first_sort_rank[mult3+1];
	d_array_second_sort_index[threadID] = mult3;
}

void gpuSetDevice(int devId) { 
	hipSetDevice(devId);
	return;
}
int gpuBlockSort(UChar *block, UInt32 *order, UInt32 *orderFirstSort, UInt32 *orderSecondSort, UInt32 *orderFirstSortRank, Int32 blockSize, Int32* sortingDepth) { 

	int limit = 64; 
	int length = blockSize;
	int originalLength = blockSize; 


	hipMalloc((unsigned int **)&global_int_original_string, sizeof(unsigned int)*originalLength);
	hipMalloc((unsigned int **)&global_first_sort_rank, sizeof(unsigned int)*originalLength);

	unsigned char *d_original_string_in; 
	hipMalloc((unsigned char **)&d_original_string_in, sizeof(unsigned char)*originalLength);
	hipMemcpy(d_original_string_in, block, sizeof(unsigned char)*originalLength, hipMemcpyHostToDevice); 
	
	int numBlocks1 = 1;
	int numThreadsPerBlock1 = originalLength/numBlocks1;

	if(numThreadsPerBlock1 > MAX_THREADS_PER_BLOCK) { 
		numBlocks1 = (int)ceil(originalLength/(float)MAX_THREADS_PER_BLOCK);
		numThreadsPerBlock1 = MAX_THREADS_PER_BLOCK;
	}
	dim3 grid1(numBlocks1, 1, 1);
	dim3 threads1(numThreadsPerBlock1, 1, 1); 


	int firstSortLength =  2*((originalLength-1)/3) + ((originalLength-1)%3);
	int secondSortLength = originalLength - firstSortLength;

	int includeLast = 0;
	if(originalLength % 3 == 1) { 
		includeLast = 1;
		firstSortLength++;
		secondSortLength--;
	}

  	thrust::device_vector<unsigned int> d_stencil(firstSortLength, 0);
	thrust::device_vector<unsigned int> d_index(firstSortLength);
	thrust::device_vector<unsigned int> d_final_index(firstSortLength);

	unsigned int *d_array_index_out = thrust::raw_pointer_cast(&d_index[0]);
	unsigned int *d_array_string_out = thrust::raw_pointer_cast(&d_stencil[0]);

	hipDeviceSynchronize();
	pack4CharsToInt<<<grid1, threads1, 0>>>(d_original_string_in, global_int_original_string, d_array_string_out, d_array_index_out, global_first_sort_rank, originalLength);
	hipDeviceSynchronize();

	if(includeLast == 1) { 
		int lastIndex = originalLength - 1;
		*(d_index.end() - 1) = lastIndex;
		*(d_stencil.end() - 1) =  (((unsigned int)block[lastIndex]) << 24) + 
				  (((unsigned int)block[(lastIndex+1) % length ]) << 16) +  
                                  (((unsigned int)block[(lastIndex+2) % length ]) << 8) + 
				  (((unsigned int)block[(lastIndex+3) % length ]));

	}

	length = firstSortLength;

#ifdef __DEBUG__
	printf("First sorting problem length : %d\n", firstSortLength);
	thrust::device_vector<unsigned int>::iterator d_index_itr = d_index.begin();
	thrust::device_vector<unsigned int>::iterator d_stencil_itr = d_stencil.begin();

	for(d_index_itr = d_index.begin(); d_index_itr != d_index.end(); ++d_index_itr) { 
		std::cout << "( " << *d_index_itr << " , " << *d_stencil_itr << " ) ";
		++d_stencil_itr;
	}
	std::cout << std::endl; 
#endif


	thrust::device_vector<unsigned int> d_string(length); 
	thrust::copy(d_stencil.begin(), d_stencil.begin() + length, d_string.begin());

	thrust::device_vector<unsigned int> d_static_index(length);
	thrust::sequence(d_static_index.begin(), d_static_index.end());

	thrust::device_vector<unsigned int> d_segment(length, 0);
	thrust::device_vector<unsigned int> d_map(length, 0);
        thrust::device_vector<unsigned int> d_integer_arr(length, 0);



	int sequenceCount = 0;

	for(sequenceCount=0; sequenceCount <= limit; sequenceCount+=4) { 
		//Changed to sort by key, this ignores the index sorting

		thrust::sort_by_key(
				thrust::make_zip_iterator( thrust::make_tuple(d_segment.begin(), d_string.begin())),
				thrust::make_zip_iterator( thrust::make_tuple(d_segment.begin() + length, d_string.begin() + length)),
				d_index.begin()
			); 
 
 
		unsigned int *d_array_string = thrust::raw_pointer_cast(&d_string[0]); 
		unsigned int *d_array_index = thrust::raw_pointer_cast(&d_index[0]);
		unsigned int *d_array_segment = thrust::raw_pointer_cast(&d_segment[0]); 
 	  	unsigned int *d_array_static_index = thrust::raw_pointer_cast(&d_static_index[0]); 
          	unsigned int *d_array_map = thrust::raw_pointer_cast(&d_map[0]); 
		unsigned int *d_array_stencil = thrust::raw_pointer_cast(&d_stencil[0]); 
		unsigned int *d_array_final_index = thrust::raw_pointer_cast(&d_final_index[0]);

		int numBlocks = 1;
		int numThreadsPerBlock = length/numBlocks;

		if(numThreadsPerBlock > MAX_THREADS_PER_BLOCK) { 
			numBlocks = (int)ceil(numThreadsPerBlock/(float)MAX_THREADS_PER_BLOCK);
			numThreadsPerBlock = MAX_THREADS_PER_BLOCK;
		}
		dim3 grid(numBlocks, 1, 1);
		dim3 threads(numThreadsPerBlock, 1, 1); 

          	hipDeviceSynchronize();
	
		findSuccessor<<<grid, threads, 0>>>(global_int_original_string, d_array_string, d_array_index, d_array_segment, d_array_stencil, d_array_map, length, originalLength, sequenceCount);
	
	        hipDeviceSynchronize();


	        thrust::copy(d_stencil.begin(), d_stencil.begin() + length, d_string.begin());

	        thrust::inclusive_scan(d_map.begin(),d_map.begin() + length, d_segment.begin());

	        hipDeviceSynchronize();
 
	        eliminateSizeOneKernel1<<<grid, threads, 0>>>( global_int_original_string, d_array_final_index, d_array_index, d_array_static_index, d_array_map, d_array_stencil, global_first_sort_rank, sequenceCount, length, originalLength);
 
                hipDeviceSynchronize();
		
		thrust::exclusive_scan(d_stencil.begin(), d_stencil.begin() + length, d_map.begin());

		thrust::scatter_if(d_segment.begin(), d_segment.begin() + length, d_map.begin(), d_stencil.begin(), d_integer_arr.begin());
		thrust::copy(d_integer_arr.begin(), d_integer_arr.begin() + length, d_segment.begin()); 

		thrust::scatter_if(d_string.begin(), d_string.begin() + length, d_map.begin(), d_stencil.begin(), d_integer_arr.begin()); 
		thrust::copy(d_integer_arr.begin(), d_integer_arr.begin() + length, d_string.begin());


		thrust::scatter_if(d_index.begin(), d_index.begin() + length, d_map.begin(), d_stencil.begin(), d_integer_arr.begin()); 
		thrust::copy(d_integer_arr.begin(), d_integer_arr.begin() + length, d_index.begin()); 

		thrust::scatter_if(d_static_index.begin(), d_static_index.begin() + length, d_map.begin(), d_stencil.begin(), d_integer_arr.begin()); 
		thrust::copy(d_integer_arr.begin(), d_integer_arr.begin() + length, d_static_index.begin()); 

		length = *(d_map.begin() + length - 1) + *(d_stencil.begin() + length - 1); 
	        if(length == 0) {
			 *sortingDepth = sequenceCount;  
             	         break;
	  	} 

	}


	if(length!=0) { 

		int size = length; 
		length = limit*2;
		int offset = limit;  
		for(offset = limit; offset < originalLength; offset+=(length/2)) {  

			thrust::sort(
					thrust::make_zip_iterator( thrust::make_tuple(d_segment.begin(), d_index.begin())),
					thrust::make_zip_iterator( thrust::make_tuple(d_segment.begin() + size, d_index.begin() + size)),
					Bar(offset, length, originalLength)
				    );

			unsigned int *d_array_index = thrust::raw_pointer_cast(&d_index[0]);
			unsigned int *d_array_segment = thrust::raw_pointer_cast(&d_segment[0]); 
			unsigned int *d_array_map = thrust::raw_pointer_cast(&d_map[0]);
			unsigned int *d_array_static_index = thrust::raw_pointer_cast(&d_static_index[0]);  
			unsigned int *d_array_stencil = thrust::raw_pointer_cast(&d_stencil[0]); 
			unsigned int *d_array_final_index = thrust::raw_pointer_cast(&d_final_index[0]);

			int numBlocks = 1;
			int numThreadsPerBlock = size/numBlocks;

			if(numThreadsPerBlock > MAX_THREADS_PER_BLOCK) { 
				numBlocks = (int)ceil(size/(float)MAX_THREADS_PER_BLOCK);
				numThreadsPerBlock = MAX_THREADS_PER_BLOCK;
			}
			dim3 grid(numBlocks, 1, 1);
			dim3 threads(numThreadsPerBlock, 1, 1); 

			hipDeviceSynchronize();

			updateSegments<<<grid, threads, 0>>>(global_int_original_string, d_array_index, d_array_segment, d_array_map, size, offset, length, originalLength);

			hipDeviceSynchronize();

			thrust::inclusive_scan(d_map.begin(), d_map.begin() + size, d_segment.begin());

			hipDeviceSynchronize();

			eliminateSizeOne<<<grid, threads, 0>>>( d_array_final_index, d_array_index, d_array_static_index, d_array_map, d_array_stencil, global_first_sort_rank, size, originalLength);
			hipDeviceSynchronize();

			thrust::exclusive_scan(d_stencil.begin(), d_stencil.begin() + size, d_map.begin());

			thrust::scatter_if(d_segment.begin(), d_segment.begin() + size, d_map.begin(), d_stencil.begin(), d_integer_arr.begin());
			thrust::copy(d_integer_arr.begin(), d_integer_arr.begin() + size, d_segment.begin()); 

			thrust::scatter_if(d_index.begin(), d_index.begin() + size, d_map.begin(), d_stencil.begin(), d_integer_arr.begin()); 
			thrust::copy(d_integer_arr.begin(), d_integer_arr.begin() + size, d_index.begin()); 

			thrust::scatter_if(d_static_index.begin(), d_static_index.begin() + size, d_map.begin(), d_stencil.begin(), d_integer_arr.begin()); 
			thrust::copy(d_integer_arr.begin(), d_integer_arr.begin() + size, d_static_index.begin()); 

			size = *(d_map.begin() + size - 1) + *(d_stencil.begin() + size - 1); 

			if(size == 0) {
				*sortingDepth = offset;
				break;
			}

			length*=2; 
		}

	}

#ifdef __DEBUG__

	printf("First Sort\n");
	thrust::device_vector<unsigned int>::iterator d_final_index_itr = d_final_index.begin();
	for(d_final_index_itr = d_final_index.begin(); d_final_index_itr != d_final_index.end(); ++d_final_index_itr) { 
		std::cout << *d_final_index_itr << " ";
	}
	std::cout << std::endl;
	std::cout << "First Sort Length " << firstSortLength << std::endl;
#endif

	int numBlocks = 1;
	int numThreadsPerBlock = secondSortLength/numBlocks;

	if(numThreadsPerBlock > MAX_THREADS_PER_BLOCK) { 
		numBlocks = (int)ceil(numThreadsPerBlock/(float)MAX_THREADS_PER_BLOCK);
		numThreadsPerBlock = MAX_THREADS_PER_BLOCK;
	}
	dim3 grid(numBlocks, 1, 1);
	dim3 threads(numThreadsPerBlock, 1, 1); 

	thrust::device_vector<unsigned char> d_second_sort(secondSortLength, 0);
	thrust::device_vector<unsigned int> d_second_sort_rank(secondSortLength, 0);
	thrust::device_vector<unsigned int> d_second_sort_index(secondSortLength);

	unsigned char *d_array_second_sort = thrust::raw_pointer_cast(&d_second_sort[0]);
	unsigned int *d_array_second_sort_rank = thrust::raw_pointer_cast(&d_second_sort_rank[0]); 
	unsigned int *d_array_second_sort_index = thrust::raw_pointer_cast(&d_second_sort_index[0]); 

	hipDeviceSynchronize();
	createSecondSort<<<grid, threads, 0>>>(d_original_string_in, global_first_sort_rank, d_array_second_sort, d_array_second_sort_rank, d_array_second_sort_index, secondSortLength);
	hipDeviceSynchronize();

	thrust::sort_by_key(
		thrust::make_zip_iterator( thrust::make_tuple(d_second_sort.begin(), d_second_sort_rank.begin())),
		thrust::make_zip_iterator( thrust::make_tuple(d_second_sort.begin() + secondSortLength, d_second_sort_rank.begin() + secondSortLength)),
		d_second_sort_index.begin()
	);  


#ifdef __DEBUG__
	printf("Second Sort\n");
	thrust::device_vector<unsigned char>::iterator d_second_sort_itr = d_second_sort.begin();
	thrust::device_vector<unsigned int>::iterator d_second_sort_rank_itr = d_second_sort_rank.begin();
	thrust::device_vector<unsigned int>::iterator d_second_sort_index_itr = d_second_sort_index.begin();
	for(d_second_sort_itr = d_second_sort.begin(); d_second_sort_itr != d_second_sort.end(); ++d_second_sort_itr) { 
		std::cout << *d_second_sort_index_itr << " ";
		++d_second_sort_rank_itr;
		++d_second_sort_index_itr;
	}
	std::cout << std::endl;
	std::cout << "Second Sort Length " << secondSortLength << std::endl;
#endif

	thrust::copy(d_final_index.begin(), d_final_index.end(), orderFirstSort);
	thrust::copy(d_second_sort_index.begin(), d_second_sort_index.end(), orderSecondSort);
	hipMemcpy(orderFirstSortRank, global_first_sort_rank, sizeof(unsigned int)*originalLength, hipMemcpyDeviceToHost); 
	
	hipFree(d_original_string_in); 
	hipFree(global_int_original_string);
	hipFree(global_first_sort_rank);
	return firstSortLength;
}
